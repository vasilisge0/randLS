#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <string>
#include "hipblas.h"
#include "hip/hip_fp16.h"
#include "magma_lapack.h"
#include "magma_v2.h"


#include "../../core/blas/blas.hpp"
#include "../../core/memory/magma_context.hpp"
#include "../../core/memory/memory.hpp"


#define CUDA_MAX_NUM_THREADS_PER_BLOCK 1024
#define CUDA_MAX_NUM_THREADS_PER_BLOCK_2D 32


namespace rls {
namespace cuda {


__host__ void generate_gaussian_sketch(magma_int_t num_rows,
                                       magma_int_t num_cols, double* sketch_mtx,
                                       hiprandGenerator_t rand_generator)
{
    hiprandGenerateNormalDouble(rand_generator, sketch_mtx, num_rows * num_cols,
                               0, 1);
    hipDeviceSynchronize();
}

__host__ void generate_gaussian_sketch(magma_int_t num_rows,
                                       magma_int_t num_cols, float* sketch_mtx,
                                       hiprandGenerator_t rand_generator)
{
    hiprandGenerateNormal(rand_generator, sketch_mtx, num_rows * num_cols, 0, 1);
    hipDeviceSynchronize();
}

template <typename index_type>
__global__ void demote_kernel(index_type num_rows, index_type num_cols,
                              double* mtx, index_type ld_mtx, __half* mtx_rp,
                              index_type ld_mtx_rp)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_rp[row + ld_mtx_rp * col] = __double2half(mtx[row + ld_mtx * col]);
    }
}

template <typename index_type>
__global__ void demote_kernel(index_type num_rows, index_type num_cols,
                              double* mtx, index_type ld_mtx, float* mtx_rp,
                              index_type ld_mtx_rp)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_rp[row + ld_mtx_rp * col] = (float)(mtx[row + ld_mtx * col]);
    }
}

template <typename index_type>
__global__ void demote_kernel(index_type num_rows, index_type num_cols,
                              float* mtx, index_type ld_mtx, __half* mtx_rp,
                              index_type ld_mtx_rp)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_rp[row + ld_mtx_rp * col] = __float2half(mtx[row + ld_mtx * col]);
    }
}

template <typename index_type>
__global__ void demote_kernel(index_type num_rows, index_type num_cols,
                              double* mtx, index_type ld_mtx, double* mtx_rp,
                              index_type ld_mtx_rp) {}

template <typename index_type>
__global__ void demote_kernel(index_type num_rows, index_type num_cols,
                              float* mtx, index_type ld_mtx, float* mtx_rp,
                              index_type ld_mtx_rp) {}

template <typename index_type>
__global__ void promote_kernel(index_type num_rows, index_type num_cols,
                               double* mtx, index_type ld_mtx, double* mtx_ip,
                               index_type ld_mtx_ip) {}

template <typename index_type>
__global__ void promote_kernel(index_type num_rows, index_type num_cols,
                               float* mtx, index_type ld_mtx, float* mtx_ip,
                               index_type ld_mtx_ip) {}

template <typename index_type>
__global__ void promote_kernel(index_type num_rows, index_type num_cols,
                               __half* mtx, index_type ld_mtx, double* mtx_ip,
                               index_type ld_mtx_ip)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_ip[row + ld_mtx_ip * col] =
            (double)__half2float(mtx[row + ld_mtx * col]);
    }
}

template <typename index_type>
__global__ void promote_kernel(index_type num_rows, index_type num_cols,
                               float* mtx, index_type ld_mtx, double* mtx_ip,
                               index_type ld_mtx_ip)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_ip[row + ld_mtx_ip * col] = (double)(mtx[row + ld_mtx * col]);
    }
}

template <typename index_type>
__global__ void promote_kernel(index_type num_rows, index_type num_cols,
                               __half* mtx, index_type ld_mtx, float* mtx_ip,
                               index_type ld_mtx_ip)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_ip[row + ld_mtx_ip * col] = __half2float(mtx[row + ld_mtx * col]);
    }
}

template <typename value_type_in, typename value_type, typename index_type>
__host__ void demote(index_type num_rows, index_type num_cols, value_type* mtx,
                     index_type ld_mtx, value_type_in* mtx_rp,
                     index_type ld_mtx_rp)
{
    int num_threads = CUDA_MAX_NUM_THREADS_PER_BLOCK_2D;
    dim3 threads_per_block(num_threads, num_threads);
    dim3 num_blocks((num_rows + threads_per_block.x - 1) / threads_per_block.x,
                    (num_cols + threads_per_block.y - 1) / threads_per_block.y);
    demote_kernel<<<num_blocks, threads_per_block>>>(num_rows, num_cols, mtx,
                                                     ld_mtx, mtx_rp, ld_mtx_rp);
    hipDeviceSynchronize();
}

template <typename value_type_in, typename value_type, typename index_type>
__host__ void promote(index_type num_rows, index_type num_cols,
                      value_type_in* mtx, index_type ld_mtx, value_type* mtx_ip,
                      index_type ld_mtx_ip)
{
    index_type num_threads = CUDA_MAX_NUM_THREADS_PER_BLOCK_2D;
    dim3 threads_per_block(num_threads, num_threads);
    dim3 num_blocks((num_rows + threads_per_block.x - 1) / threads_per_block.x,
                    (num_cols + threads_per_block.y - 1) / threads_per_block.y);
    promote_kernel<<<num_blocks, threads_per_block>>>(
        num_rows, num_cols, mtx, ld_mtx, mtx_ip, ld_mtx_ip);
    hipDeviceSynchronize();
}


template __global__ void demote_kernel(magma_int_t num_rows,
                                       magma_int_t num_cols, double* mtx,
                                       magma_int_t ld_mtx, __half* mtx_rp,
                                       magma_int_t ld_mtx_rp);
template __global__ void demote_kernel(magma_int_t num_rows,
                                       magma_int_t num_cols, float* mtx,
                                       magma_int_t ld_mtx, __half* mtx_rp,
                                       magma_int_t ld_mtx_rp);
template __global__ void demote_kernel(magma_int_t num_rows,
                                       magma_int_t num_cols, float* mtx,
                                       magma_int_t ld_mtx, float* mtx_rp,
                                       magma_int_t ld_mtx_rp);
template __global__ void demote_kernel(magma_int_t num_rows,
                                       magma_int_t num_cols, double* mtx,
                                       magma_int_t ld_mtx, double* mtx_rp,
                                       magma_int_t ld_mtx_rp);
template __global__ void promote_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, __half* mtx,
                                        magma_int_t ld_mtx, double* mtx_ip,
                                        magma_int_t ld_mtx_ip);
template __global__ void promote_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, float* mtx,
                                        magma_int_t ld_mtx, double* mtx_ip,
                                        magma_int_t ld_mtx_ip);

template __host__ void demote(magma_int_t num_rows, magma_int_t num_cols,
                              double* mtx, magma_int_t ld_mtx, __half* mtx_rp,
                              magma_int_t ld_mtx_rp);
template __host__ void demote(magma_int_t num_rows, magma_int_t num_cols,
                              float* mtx, magma_int_t ld_mtx, __half* mtx_rp,
                              magma_int_t ld_mtx_rp);
template __host__ void demote(magma_int_t num_rows, magma_int_t num_cols,
                              double* mtx, magma_int_t ld_mtx, float* mtx_rp,
                              magma_int_t ld_mtx_rp);
template __host__ void promote(magma_int_t num_rows, magma_int_t num_cols,
                               __half* mtx, magma_int_t ld_mtx, double* mtx_ip,
                               magma_int_t ld_mtx_ip);
template __host__ void promote(magma_int_t num_rows, magma_int_t num_cols,
                               __half* mtx, magma_int_t ld_mtx, float* mtx_ip,
                               magma_int_t ld_mtx_ip);
template __host__ void promote(magma_int_t num_rows, magma_int_t num_cols,
                               float* mtx, magma_int_t ld_mtx, double* mtx_ip,
                               magma_int_t ld_mtx_ip);

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              double* mtx_in, index_type ld_in, __half* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = __double2half(mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              double* mtx_in, index_type ld_in, float* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = (float)(mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              float* mtx_in, index_type ld_in, __half* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = __float2half(mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              __half* mtx_in, index_type ld_in, __half* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = __float2half(mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              __half* mtx_in, index_type ld_in, double* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = __half2float(mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              __half* mtx_in, index_type ld_in, float* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = __half2float(mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              float* mtx_in, index_type ld_in, double* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = (double)(mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              double* mtx_in, index_type ld_in, double* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = (mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              int* mtx_in, index_type ld_in, int* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = (mtx_in[row + ld_in * col]);
    }
}

template <typename index_type>
__global__ void convert_kernel(index_type num_rows, index_type num_cols,
                              float* mtx_in, index_type ld_in, float* mtx_out,
                              index_type ld_out)
{
    auto row = blockIdx.x * blockDim.x + threadIdx.x;
    auto col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[row + ld_out * col] = (mtx_in[row + ld_in * col]);
        if (isnan(mtx_out[row + ld_out * col])) {
            printf("-----> nan\n");
        }
    }
}

template __global__ void convert_kernel(magma_int_t num_rows,
                                       magma_int_t num_cols, double* mtx_in,
                                       magma_int_t ld_in, __half* mtx_out,
                                       magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, float* mtx_in,
                                        magma_int_t ld_in, __half* mtx_out,
                                        magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, double* mtx_in,
                                        magma_int_t ld_in, float* mtx_out,
                                        magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, __half* mtx_in,
                                        magma_int_t ld_in, float* mtx_out,
                                        magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, __half* mtx_in,
                                        magma_int_t ld_in, double* mtx_out,
                                        magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, float* mtx_in,
                                        magma_int_t ld_in, double* mtx_out,
                                        magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, double* mtx_in,
                                        magma_int_t ld_in, double* mtx_out,
                                        magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, float* mtx_in,
                                        magma_int_t ld_in, float* mtx_out,
                                        magma_int_t ld_out);
template __global__ void convert_kernel(magma_int_t num_rows,
                                        magma_int_t num_cols, int* mtx_in,
                                        magma_int_t ld_in, int* mtx_out,
                                        magma_int_t ld_out);

template <typename value_type_in, typename value_type_out, typename index_type>
__host__ void convert(index_type num_rows, index_type num_cols,
                      value_type_in* mtx_in, index_type ld_in, value_type_out* mtx_out,
                      index_type ld_out)
{
    index_type num_threads = CUDA_MAX_NUM_THREADS_PER_BLOCK_2D;
    dim3 threads_per_block(num_threads, num_threads);
    dim3 num_blocks((num_rows + threads_per_block.x - 1) / threads_per_block.x,
                    (num_cols + threads_per_block.y - 1) / threads_per_block.y);
    convert_kernel<<<num_blocks, threads_per_block>>>(
        num_rows, num_cols, mtx_in, ld_in, mtx_out, ld_out);
    hipDeviceSynchronize();
}

template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      double* mtx_in, magma_int_t ld_in, float* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      double* mtx_in, magma_int_t ld_in, double* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      double* mtx_in, magma_int_t ld_in, __half* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      float* mtx_in, magma_int_t ld_in, __half* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      __half* mtx_in, magma_int_t ld_in, float* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      __half* mtx_in, magma_int_t ld_in, double* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      __half* mtx_in, magma_int_t ld_in, __half* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      float* mtx_in, magma_int_t ld_in, double* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert(magma_int_t num_rows, magma_int_t num_cols,
                      float* mtx_in, magma_int_t ld_in, float* mtx_out,
                      magma_int_t ld_out);
template __host__  void convert<int, int, int>(int num_rows, int num_cols,
                      int* mtx_in, int ld_in, int* mtx_out,
                      int ld_out);

template <typename value_type, typename index_type>
__global__ void transpose_kernel(index_type num_rows, index_type num_cols, value_type* mtx_in, index_type ld_in, value_type* mtx_out, index_type ld_out)
{
    auto row = blockIdx.y * blockDim.y + threadIdx.y;
    auto col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < num_rows) && (col < num_cols)) {
        mtx_out[col + ld_out*row] = mtx_in[row + ld_in*col];
    }
}

template __global__ void transpose_kernel(magma_int_t num_rows, magma_int_t num_cols, double* mtx_in, magma_int_t ld_in, double* mtx_out, magma_int_t ld_out);
template __global__ void transpose_kernel(magma_int_t num_rows, magma_int_t num_cols, float* mtx_in, magma_int_t ld_in, float* mtx_out, magma_int_t ld_out);
template __global__ void transpose_kernel(magma_int_t num_rows, magma_int_t num_cols, __half* mtx_in, magma_int_t ld_in, __half* mtx_out, magma_int_t ld_out);

template <typename value_type, typename index_type>
__host__ void transpose(index_type num_rows, index_type num_cols, value_type* mtx_in, index_type ld_in, value_type* mtx_out, index_type ld_out)
{
    index_type num_threads = CUDA_MAX_NUM_THREADS_PER_BLOCK_2D;
    dim3 threads_per_block(num_threads, num_threads);
    dim3 num_blocks((num_cols + threads_per_block.x - 1) / threads_per_block.x,
                    (num_rows + threads_per_block.y - 1) / threads_per_block.y);
    transpose_kernel<<<num_blocks, threads_per_block>>>(num_rows, num_cols, mtx_in, ld_in, mtx_out, ld_out);
    hipDeviceSynchronize();
}

template __host__ void transpose(magma_int_t num_rows, magma_int_t num_cols, double* mtx_in, magma_int_t ld_in, double* mtx_out, magma_int_t ld_out);
template __host__ void transpose(magma_int_t num_rows, magma_int_t num_cols, float* mtx_in, magma_int_t ld_in, float* mtx_out, magma_int_t ld_out);
template __host__ void transpose(magma_int_t num_rows, magma_int_t num_cols, __half* mtx_in, magma_int_t ld_in, __half* mtx_out, magma_int_t ld_out);


}  // namespace cuda
}  // namespace rls
